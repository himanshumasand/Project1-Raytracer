#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include <cutil_math.h>
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
  
	ray r;
	glm::vec3 a, b, m, h, v, p;
	
	//r.origin = glm::vec3(0,0,0);
	//r.direction = glm::vec3(0,0,-1);
	
	a = glm::cross(view, up);
	b = glm::cross(a, view);
	m = eye + view;
	h.x = 0.5f; //a * view.length() * fov.x / a.length();
	v.y = 0.5f; 

	p = glm::vec3((float)(m.x + ((((2.0*x)/(resolution.x-1))-1)*h.x)+((((2.0*y)/(resolution.y-1))-1)*v.x)), (float)(m.y + ((((2.0*x)/(resolution.x-1))-1)*h.y)+((((2.0*y)/(resolution.y-1))-1)*v.y)), (float)(m.z + ((((2.0*x)/(resolution.x-1))-1)*h.z)+((((2.0*y)/(resolution.y-1))-1)*v.z)));
		
	r.origin = eye;
	r.direction = p - eye;
	float mag = (p - eye).length();
	r.direction = r.direction / mag;
	
	return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors, 
                            staticGeom* geoms, int numberOfGeoms, material* mats, int numberOfMaterials){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = resolution.x * resolution.y - ( x + (y * resolution.x));
  float dist, distLight;
  float minDist = 99999, minDistLight = 99999;
  int indexOfGeom;
  ray raycast;
  glm::vec3 intersectionPoint, normal;

  glm::vec3 lpos;
  
  //gets the position of the last defined light source
  for(int i = 0; i < numberOfGeoms; i++)
  {
	  if(mats[geoms[i].materialid].emittance > 1)
	  {
		  lpos = geoms[i].translation;
	  }
  }

  //lpos = glm::vec3(0,9,0);
  glm::vec3 tempColor, lightDir;
  float kAmbient = 0.4f, kDiffuse = 0.4f;

  if((x<=resolution.x && y<=resolution.y)){

	  raycast = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov);
	  for(int i = 0; i < numberOfGeoms; i++)
	  {		  
		  if(geoms[i].type == SPHERE)
			  dist = sphereIntersectionTest( geoms[i] , raycast, intersectionPoint, normal);
		  else if(geoms[i].type == CUBE)
			  dist = boxIntersectionTest( geoms[i], raycast, intersectionPoint, normal);

		  if(dist != -1 && dist < minDist)
		  {  
			  minDist = dist;
			  indexOfGeom = i;
		  }
	  }

	  if(minDist == 99999)
	  {
		  colors[index] = glm::vec3(0,0,0);
	  }

	  else
	  {
		  tempColor = mats[geoms[indexOfGeom].materialid].color;
		  colors[index] = kAmbient * tempColor;
		  
		  ray lightRay;
		  lightDir = lpos - intersectionPoint;
		  lightDir = glm::normalize(lightDir);
		  float factor = glm::dot(normal, lightDir);
		  lightRay.origin = intersectionPoint;
		  lightRay.direction = lightDir;
		  distLight = -1;
		  minDistLight = 99999;
		  
		 // if(geoms[indexOfGeom].type == SPHERE)
			//  distLight = sphereIntersectionTest( geoms[indexOfGeom] , lightRay, intersectionPoint, normal);
		 // else if(geoms[indexOfGeom].type == CUBE)
			//  distLight = boxIntersectionTest( geoms[indexOfGeom], lightRay, intersectionPoint, normal);

		 // if(distLight != -1 && distLight < minDistLight)
			//{  
			//	minDistLight = distLight;
			//}

		  for(int i = 0; i < numberOfGeoms - 1; i++)
		  {
			  if(i != 2)
			  {
				  if(geoms[i].type == SPHERE)
					  distLight = sphereIntersectionTest( geoms[i] , lightRay, intersectionPoint, normal);
				  else if(geoms[i].type == CUBE)
					  distLight = boxIntersectionTest( geoms[i], lightRay, intersectionPoint, normal);

				  if(distLight != -1 && distLight < minDistLight)
				  {  
					  minDistLight = distLight;
				  }
			  }
		  }

		  if(minDistLight == 99999)
		  {
			  colors[index] += kDiffuse * factor;
		  }

		  //clamp(colors[index].x, 0.0f, 1.0f);
		  //clamp(colors[index].y, 0.0f, 1.0f);
		  //clamp(colors[index].z, 0.0f, 1.0f);
	  }
      //colors[index] = glm::vec3(1,0,0);// generateRandomNumberFromThread(resolution, time, x, y);
   }
}


//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  material* matList = new material[numberOfMaterials];
  for(int i = 0; i < numberOfMaterials; i++)
  {
	  material newMaterial;
	  newMaterial.absorptionCoefficient = materials[i].absorptionCoefficient;
	  newMaterial.color = materials[i].color;
	  newMaterial.emittance = materials[i].emittance;
	  newMaterial.hasReflective = materials[i].hasReflective;
	  newMaterial.hasRefractive = materials[i].hasRefractive;
	  newMaterial.hasScatter = materials[i].hasScatter;
	  newMaterial.indexOfRefraction = materials[i].indexOfRefraction;
	  newMaterial.reducedScatterCoefficient = materials[i].reducedScatterCoefficient;
	  newMaterial.specularColor = materials[i].specularColor;
	  newMaterial.specularExponent = materials[i].specularExponent;
	  matList[i] = newMaterial;
  }

  material* cudamats = NULL;
  hipMalloc((void**)&cudamats, numberOfMaterials*sizeof(material));
  hipMemcpy( cudamats, matList, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

  //kernel launches
  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, cudamats, numberOfMaterials);

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  delete geomList;
  hipFree( cudamats );
  delete matList;

  // make certain the kernel has completed 
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}
